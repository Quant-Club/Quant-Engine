#include <hip/hip_runtime.h>

#include <math.h>

extern "C" {

__global__ void moving_average_kernel(const double* prices,
                                    double* result,
                                    int period,
                                    int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size - period + 1) return;

    double sum = 0.0;
    for (int i = 0; i < period; i++) {
        sum += prices[idx + i];
    }
    result[idx] = sum / period;
}

__global__ void exponential_moving_average_kernel(const double* prices,
                                                double* result,
                                                double alpha,
                                                int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return;

    if (idx == 0) {
        result[0] = prices[0];
    } else {
        result[idx] = alpha * prices[idx] + (1 - alpha) * result[idx - 1];
    }
}

__global__ void bollinger_bands_kernel(const double* prices,
                                     double* upperBand,
                                     double* middleBand,
                                     double* lowerBand,
                                     int period,
                                     double numStdDev,
                                     int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size - period + 1) return;

    // Calculate SMA
    double sum = 0.0;
    double sumSq = 0.0;
    for (int i = 0; i < period; i++) {
        double price = prices[idx + i];
        sum += price;
        sumSq += price * price;
    }
    
    double sma = sum / period;
    double variance = (sumSq - (sum * sum / period)) / (period - 1);
    double stdDev = sqrt(variance);
    
    middleBand[idx] = sma;
    upperBand[idx] = sma + numStdDev * stdDev;
    lowerBand[idx] = sma - numStdDev * stdDev;
}

__global__ void rsi_kernel(const double* prices,
                          double* rsi,
                          int period,
                          int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size - period) return;

    double sumGain = 0.0;
    double sumLoss = 0.0;

    // Calculate initial gains and losses
    for (int i = 1; i <= period; i++) {
        double diff = prices[idx + i] - prices[idx + i - 1];
        if (diff > 0) {
            sumGain += diff;
        } else {
            sumLoss -= diff;
        }
    }

    double avgGain = sumGain / period;
    double avgLoss = sumLoss / period;
    
    if (avgLoss == 0.0) {
        rsi[idx] = 100.0;
    } else {
        double rs = avgGain / avgLoss;
        rsi[idx] = 100.0 - (100.0 / (1.0 + rs));
    }
}

} // extern "C"
