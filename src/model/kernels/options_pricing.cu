#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include <hiprand/hiprand_kernel.h>

extern "C" {

__device__ double normalCDF(double x) {
    return 0.5 * (1.0 + erf(x / sqrt(2.0)));
}

__global__ void black_scholes_kernel(const OptionData* options,
                                   double* callPrices,
                                   double* putPrices,
                                   int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return;

    const OptionData& opt = options[idx];
    
    double sqrtTime = sqrt(opt.timeToExpiry);
    double d1 = (log(opt.spotPrice / opt.strikePrice) + 
                (opt.riskFreeRate + 0.5 * opt.volatility * opt.volatility) * opt.timeToExpiry) / 
                (opt.volatility * sqrtTime);
    double d2 = d1 - opt.volatility * sqrtTime;

    double nd1 = normalCDF(d1);
    double nd2 = normalCDF(d2);
    double nnd1 = normalCDF(-d1);
    double nnd2 = normalCDF(-d2);

    double discountFactor = exp(-opt.riskFreeRate * opt.timeToExpiry);
    
    callPrices[idx] = opt.spotPrice * nd1 - opt.strikePrice * discountFactor * nd2;
    putPrices[idx] = opt.strikePrice * discountFactor * nnd2 - opt.spotPrice * nnd1;
}

__global__ void monte_carlo_kernel(const SimulationParams* params,
                                 double* paths,
                                 int numPaths,
                                 int numSteps,
                                 unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numPaths) return;

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    const SimulationParams& param = params[blockIdx.y];
    double dt = param.timeHorizon / numSteps;
    double sqrtDt = sqrt(dt);
    
    int baseIdx = (blockIdx.y * numPaths + idx) * numSteps;
    paths[baseIdx] = param.spotPrice;

    for (int step = 1; step < numSteps; step++) {
        double z = hiprand_normal(&state);
        double drift = (param.riskFreeRate - 0.5 * param.volatility * param.volatility) * dt;
        double diffusion = param.volatility * sqrtDt * z;
        
        paths[baseIdx + step] = paths[baseIdx + step - 1] * exp(drift + diffusion);
    }
}

} // extern "C"
