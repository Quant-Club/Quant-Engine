#include <hip/hip_runtime.h>

#include <math.h>

extern "C" {

__global__ void portfolio_optimization_kernel(const double* returns,
                                           const double* covariance,
                                           double* weights,
                                           double riskFreeRate,
                                           double targetReturn,
                                           int numAssets) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numAssets) return;

    // Simple implementation of Mean-Variance Optimization using gradient descent
    // Note: In practice, you would want to use a more sophisticated optimization algorithm
    
    const double learningRate = 0.01;
    const int maxIterations = 1000;
    const double epsilon = 1e-6;

    // Initialize weights equally
    weights[idx] = 1.0 / numAssets;
    __syncthreads();

    for (int iter = 0; iter < maxIterations; iter++) {
        // Calculate portfolio return and risk
        double portReturn = 0.0;
        double portRisk = 0.0;
        
        for (int i = 0; i < numAssets; i++) {
            portReturn += weights[i] * returns[i];
            for (int j = 0; j < numAssets; j++) {
                portRisk += weights[i] * weights[j] * covariance[i * numAssets + j];
            }
        }

        // Calculate gradients
        double returnGrad = returns[idx] - targetReturn;
        double riskGrad = 0.0;
        for (int j = 0; j < numAssets; j++) {
            riskGrad += weights[j] * covariance[idx * numAssets + j];
        }

        // Update weights
        double grad = returnGrad + riskGrad;
        weights[idx] -= learningRate * grad;

        // Project weights to satisfy constraints
        double sum = 0.0;
        for (int i = 0; i < numAssets; i++) {
            weights[i] = max(0.0, weights[i]); // Non-negativity constraint
            sum += weights[i];
        }
        weights[idx] /= sum; // Sum to 1 constraint

        __syncthreads();

        // Check convergence
        if (abs(grad) < epsilon) {
            break;
        }
    }
}

__global__ void value_at_risk_kernel(const double* returns,
                                   const double* weights,
                                   double* var,
                                   double confidence,
                                   int horizon,
                                   int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return;

    // Calculate portfolio returns
    double portReturn = 0.0;
    for (int i = 0; i < size; i++) {
        portReturn += returns[i] * weights[i];
    }

    // Sort returns (simple bubble sort for demonstration)
    // In practice, use more efficient sorting algorithms
    __shared__ double sortedReturns[1024];
    sortedReturns[idx] = returns[idx];
    __syncthreads();

    for (int i = 0; i < size - 1; i++) {
        for (int j = 0; j < size - i - 1; j++) {
            if (sortedReturns[j] > sortedReturns[j + 1]) {
                double temp = sortedReturns[j];
                sortedReturns[j] = sortedReturns[j + 1];
                sortedReturns[j + 1] = temp;
            }
        }
    }
    __syncthreads();

    // Calculate VaR
    if (idx == 0) {
        int varIndex = (int)((1.0 - confidence) * size);
        *var = -sortedReturns[varIndex] * sqrt(horizon);
    }
}

} // extern "C"
